#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>
#include <math.h>

const __int64 N = 1024; //048;
const __int64 blocksize = 16; // 2048;
const __int64 gridsize = N / blocksize;

__global__ 
void simple(float *c, float *a, float *b) 
{
	unsigned int blockId = gridDim.y * blockIdx.x + blockIdx.y;
	unsigned int index = blockId * blockDim.x * blockDim.y + blockDim.x * threadIdx.y + threadIdx.x;
	c[index] = a[index] + b[index];
}

int main()
{
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	printf("Warp size: %i \n", properties.warpSize);
	printf("Max Threads per Block: %i \n", properties.maxThreadsPerBlock);
	printf("Max Threads, x: %i, y: %i, z: %i \n", properties.maxThreadsDim[0], properties.maxThreadsDim[1], properties.maxThreadsDim[2]);
	printf("Max Grid Size, x: %i, y: %i, z: %i \n", properties.maxGridSize[0], properties.maxGridSize[1], properties.maxGridSize[2]);


	float *c = new float[N*N];
	float *a = new float[N*N];
	float *b = new float[N*N];
	float *cd, *ad, *bd;
	const __int64 size = N*N*sizeof(float);
	
	float time;

	hipEvent_t startEvent;
	hipEventCreate(&startEvent);
	

	hipEvent_t endEvent;
	hipEventCreate(&endEvent);
	


	for (__int64 j = 0; j < N; j++) {
		for (__int64 i = 0; i < N; i++) {
			a[i + j*N] = (float)i;
			b[i + j*N] = (float)j / 100000000.0f;
		}
	}
	

	hipMalloc( (void**)&cd, size );
	hipMalloc((void**)&ad, size);
	hipMalloc((void**)&bd, size);
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, blocksize);
	dim3 dimGrid( gridsize, gridsize );

	hipEventRecord(startEvent, 0);
	simple<<<dimGrid, dimBlock>>>(cd, ad, bd);
	hipEventRecord(endEvent, 0);

	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	hipFree(ad);
	hipFree(bd);
	
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&time, startEvent, endEvent);


	
	for (int j = N-8; j < N; j++) {
		for (int i = N-8; i < N; i++) {
			printf("%0.4f ", c[i + j*N]);
		}
		printf("\n");
	}
	

	printf("Time taken: %f\n", time);
	delete[] c;
	delete[] a;
	delete[] b;
	printf("done\n");
	return EXIT_SUCCESS;
}
