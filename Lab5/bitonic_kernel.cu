#include "hip/hip_runtime.h"

// This is not really C++-code but pretty plain C code, but we compile it
// as C++ so we can integrate with CUDA seamlessly.

// If you plan on submitting your solution for the Parallel Sorting Contest,
// please keep the split into main file and kernel file, so we can easily
// insert other data.

#include <stdio.h>
#include "bitonic_kernel.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__device__ static inline void exchange(int *i, int *j)
{
	int k;
	k = *i;
	*i = *j;
	*j = k;
}

// No, this is not GPU code yet but just a copy of the CPU code, but this
// is where I want to see your GPU code!

__global__ void bitonic_block(int* data, int k, int j)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int ixj=i^j; // Calculate indexing
  
  if ((ixj)>i)
  {
    if ((i&k)==0 && data[i]>data[ixj]) exchange(&data[i],&data[ixj]);
    if ((i&k)!=0 && data[i]<data[ixj]) exchange(&data[i],&data[ixj]);
  }
}


float bitonic_gpu(int *data, int N)
{
  int j,k;
  
  float timeGPU;
  
  int* devdata;
  int size = sizeof(int) * N;

  dim3 dimBlock(N / 1024, 1);
  dim3 dimGrid(1024, 1);

  hipEvent_t startEvent;
  hipEvent_t endEvent;

	hipEventCreate(&startEvent);
	hipEventCreate(&endEvent);

	hipMalloc((void**)&devdata, size);
	hipMemcpy(devdata, data, size, hipMemcpyHostToDevice);
  
	hipEventRecord(startEvent, 0);
  for (k=2;k<=N;k=2*k) // Outer loop, double size for each step
  {
    for (j=k>>1;j>0;j=j>>1) // Inner loop, half size for each step
    {
      bitonic_block<<< dimBlock, dimGrid >>>(devdata, k, j);
    }
  }
  hipEventRecord(endEvent, 0);
  
  hipEventSynchronize(endEvent);
  hipEventElapsedTime(&timeGPU, startEvent, endEvent);
  
	hipError_t err = hipPeekAtLastError();
	if (err) printf("hipPeekAtLastError %d %s\n", err, hipGetErrorString(err));

	// Only the result needs copying!
	hipMemcpy(data, devdata, size, hipMemcpyDeviceToHost);
	hipFree(devdata);
	

	
	return timeGPU;
}
